#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "tdca_types.h"
#include "tdca_globals.h"

#include <hip/hip_runtime_api.h>
#include <cuda_d3d11_interop.h>

hipGraphicsResource* g_graphics_resource;

voxel_instance_transforms* g_instances;

__device__ unsigned int cell_state(tdca* tdca, cell_index cell)
{
    if(cell < 0 || cell > tdca->lifespace.cell_count)
    {
        return 0;
    }
    else
    {
        return tdca->lifespace.last_cells[cell].state;
    }
}

__device__ bool is_in_bounds_relative(tdca* tdca, cell_index origin, cell_index target)
{
    int cells_per_axis = 1 << tdca->lifespace.subdivision_count;
    int cells_per_slice = 1 << (tdca->lifespace.subdivision_count * 2);

    // the row of cells the current cell index resides in
    cell_index row_start_cell_inclusive = origin - (origin % cells_per_axis);
    cell_index row_end_cell_inclusive = row_start_cell_inclusive + cells_per_axis - 1;

    // the vertical slice of cells the current cell resides in
    cell_index slice_start_cell_inclusive = origin - (origin % cells_per_slice);
    cell_index slice_end_cell_inclusive = slice_start_cell_inclusive + cells_per_slice - 1;

    int difference =  target - origin;

    bool cell_is_right =   difference == 1 
                            || difference == -cells_per_axis + 1 
                            || difference == cells_per_axis + 1 
                            || difference == - cells_per_slice + 1 
                            || difference == - cells_per_slice + -cells_per_axis + 1 
                            || difference == - cells_per_slice + cells_per_axis + 1 
                            || difference == + cells_per_slice + 1 
                            || difference == + cells_per_slice + -cells_per_axis + 1 
                            || difference == + cells_per_slice + cells_per_axis + 1;
    
    bool cell_is_left =    difference == -1 
                            || difference == -cells_per_axis - 1 
                            || difference == cells_per_axis - 1 
                            || difference == - cells_per_slice - 1 
                            || difference == - cells_per_slice + -cells_per_axis - 1 
                            || difference == - cells_per_slice + cells_per_axis - 1 
                            || difference == + cells_per_slice - 1 
                            || difference == + cells_per_slice + -cells_per_axis - 1 
                            || difference == + cells_per_slice + cells_per_axis - 1;

    bool cell_is_top =     difference == -cells_per_axis
                            || difference == -cells_per_axis - 1
                            || difference == -cells_per_axis + 1
                            || difference == -cells_per_axis - cells_per_slice
                            || difference == -cells_per_axis - cells_per_slice - 1
                            || difference == -cells_per_axis - cells_per_slice + 1
                            || difference == -cells_per_axis +  cells_per_slice
                            || difference == -cells_per_axis +  cells_per_slice - 1
                            || difference == -cells_per_axis +  cells_per_slice + 1;

    bool cell_is_bottom =  difference == cells_per_axis
                            || difference == cells_per_axis - 1
                            || difference == cells_per_axis + 1
                            || difference == cells_per_axis - cells_per_slice
                            || difference == cells_per_axis - cells_per_slice - 1
                            || difference == cells_per_axis - cells_per_slice + 1
                            || difference == cells_per_axis +  cells_per_slice
                            || difference == cells_per_axis +  cells_per_slice - 1
                            || difference == cells_per_axis +  cells_per_slice + 1;

    if(cell_is_right)
        if ((origin + 1) > row_end_cell_inclusive) return false;
    if(cell_is_left)
        if ((origin - 1) < row_start_cell_inclusive) return false;
    if(cell_is_top)
        if ((origin - cells_per_axis) < slice_start_cell_inclusive) return false;
    if(cell_is_bottom)
        if ((origin + cells_per_axis) > slice_end_cell_inclusive) return false;

    return true;
}

__device__ void count_alive_neighbors_moore(tdca* tdca, cell_index cell, unsigned int* alive_neighbors_count)
{
    int cells_per_axis = 1 << tdca->lifespace.subdivision_count;
    int cells_per_slice = 1 << (tdca->lifespace.subdivision_count * 2);

    if(is_in_bounds_relative(tdca, cell, cell - 1))                                       *alive_neighbors_count += (cell_state(tdca, cell - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + 1))                                       *alive_neighbors_count += (cell_state(tdca, cell + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_axis))                          *alive_neighbors_count += (cell_state(tdca, cell - cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_axis - 1))                      *alive_neighbors_count += (cell_state(tdca, cell - cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_axis + 1))                      *alive_neighbors_count += (cell_state(tdca, cell - cells_per_axis + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_axis))                          *alive_neighbors_count += (cell_state(tdca, cell + cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_axis - 1))                      *alive_neighbors_count += (cell_state(tdca, cell + cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_axis + 1))                      *alive_neighbors_count += (cell_state(tdca, cell + cells_per_axis + 1) > 0 ? 1 : 0);

    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice))                         *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice - 1))                     *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice + 1))                     *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice - cells_per_axis))        *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice - cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice - cells_per_axis - 1))    *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice - cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice - cells_per_axis + 1))    *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice - cells_per_axis + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice + cells_per_axis))        *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice + cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice + cells_per_axis - 1))    *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice + cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice + cells_per_axis + 1))    *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice + cells_per_axis + 1) > 0 ? 1 : 0);

    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice))                         *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice - 1))                     *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice + 1))                     *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice - cells_per_axis))        *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice - cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice - cells_per_axis - 1))    *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice - cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice - cells_per_axis + 1))    *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice - cells_per_axis + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice + cells_per_axis))        *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice + cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice + cells_per_axis - 1))    *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice + cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice + cells_per_axis + 1))    *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice + cells_per_axis + 1) > 0 ? 1 : 0);
}

__device__ void count_alive_neighbors_von_neumann(tdca* tdca, cell_index cell, unsigned int* alive_neighbors_count)
{
    int cells_per_axis = 1 << tdca->lifespace.subdivision_count;
    int cells_per_slice = 1 << (tdca->lifespace.subdivision_count * 2);

    if(is_in_bounds_relative(tdca, cell, cell - 1))                                       *alive_neighbors_count += (cell_state(tdca, cell - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + 1))                                       *alive_neighbors_count += (cell_state(tdca, cell + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_axis))                          *alive_neighbors_count += (cell_state(tdca, cell - cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_axis))                          *alive_neighbors_count += (cell_state(tdca, cell + cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice))                         *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice))                         *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice) > 0 ? 1 : 0);
}

__global__ void update_lifespace_partition(tdca* tdca)
{
    unsigned int thread_id = threadIdx.x;
    unsigned int block_id = blockIdx.x;
    unsigned int number_of_threads_per_block = blockDim.x;
    unsigned int number_of_blocks_per_grid = gridDim.x;

    int cells_per_thread = tdca->lifespace.cell_count / (number_of_threads_per_block * number_of_blocks_per_grid);
    
    cell_index starting_cell = (block_id * (cells_per_thread * number_of_threads_per_block)) + (thread_id * cells_per_thread);
    cell_index ending_cell = starting_cell + cells_per_thread - 1;
    
    for(cell_index cell = starting_cell; cell < ending_cell + 1; cell++)
    {
        unsigned int alive_neighbor_count = 0;
        if(tdca->rule.neighborhood == tdca_rule::MOORE)
        {
            count_alive_neighbors_moore(tdca, cell, &alive_neighbor_count);
        }
        else if(tdca->rule.neighborhood == tdca_rule::VON_NEUMANN)
        {
            count_alive_neighbors_von_neumann(tdca, cell, &alive_neighbor_count);
        }

        if(cell_state(tdca, cell) == tdca_cell::ALIVE)
        {
            if(tdca->rule.necessary_amounts_of_alive_neighbors_for_surviving[alive_neighbor_count] == 0)
            {
                if(tdca->rule.state_count == 2)
                {
                    tdca->lifespace.current_cells[cell].state = tdca_cell::DEAD;
                    tdca->lifespace.current_cells[cell].lifetime = 0;
                }
                else
                {
                    tdca->lifespace.current_cells[cell].state = tdca_cell::DYING;
                    tdca->lifespace.current_cells[cell].lifetime--;
                }
            }
        }
        else if(cell_state(tdca, cell) == tdca_cell::DYING)
        {
            if(tdca->lifespace.last_cells[cell].lifetime == 0)
            {
                tdca->lifespace.current_cells[cell].state = tdca_cell::DEAD;
            }
            else
            {
                tdca->lifespace.current_cells[cell].lifetime--;
            }
        }
        else if(cell_state(tdca, cell) == tdca_cell::DEAD)
        {
            if(tdca->rule.necessary_amounts_of_alive_neighbors_for_birth[alive_neighbor_count] == 1)
            {
                tdca->lifespace.current_cells[cell].state = tdca_cell::ALIVE;
                tdca->lifespace.current_cells[cell].lifetime = tdca->rule.state_count - 1;
            }
        }
    }
}

void cuda_update_current_buffer(tdca* tdca)
{
    hipMemcpy(tdca->lifespace.current_cells, tdca->lifespace.last_cells, sizeof(tdca_cell) * tdca->lifespace.cell_count, hipMemcpyDefault);

    unsigned int partition_count = 1 << tdca->spacial_partitioning_scheme.subdivision_count;

    update_lifespace_partition<<<partition_count,1024>>>(tdca);
    int error_code = hipDeviceSynchronize();
}

void cuda_init(tdca** tdca, unsigned int tdca_size_in_bytes)
{
    hipMallocManaged(tdca, tdca_size_in_bytes);
    (*tdca)->lifespace.subdivision_count = 6;
    // (*tdca)->lifespace.subdivision_count = 7;
    // (*tdca)->lifespace.subdivision_count = 8;
    // (*tdca)->lifespace.subdivision_count = 9;
    (*tdca)->lifespace.cell_count = 1 << ((*tdca)->lifespace.subdivision_count * 3);

    hipMallocManaged(&(*tdca)->lifespace.current_cells, (*tdca)->lifespace.cell_count * sizeof(tdca_cell));
    hipMemset(&(*tdca)->lifespace.current_cells, 0, (*tdca)->lifespace.cell_count * sizeof(tdca_cell));

    hipMallocManaged(&(*tdca)->lifespace.last_cells, (*tdca)->lifespace.cell_count * sizeof(tdca_cell));
    hipMemset(&(*tdca)->lifespace.last_cells, 0, (*tdca)->lifespace.cell_count * sizeof(tdca_cell));

    (*tdca)->rule.neighborhood = (*tdca)->rule.neighborhood::MOORE;

    (*tdca)->spacial_partitioning_scheme.scheme = (*tdca)->spacial_partitioning_scheme.scheme::BINARY;
    (*tdca)->spacial_partitioning_scheme.subdivision_count = 8;
    // (*tdca)->spacial_partitioning_scheme.subdivision_count = 11;
    // (*tdca)->spacial_partitioning_scheme.subdivision_count = 14;
    // (*tdca)->spacial_partitioning_scheme.subdivision_count = 17;

    hipMallocManaged(&g_instances, (*tdca)->lifespace.cell_count * sizeof(voxel_instance_transforms));
    hipMemset(g_instances, 0, (*tdca)->lifespace.cell_count * sizeof(voxel_instance_transforms));
}

void cuda_free(tdca* tdca)
{
    hipFree(tdca->lifespace.current_cells);
    hipFree(tdca->lifespace.last_cells);
    hipFree(tdca);
    hipFree(g_dx11_voxel);
    hipFree(g_instances);
}

DirectX::XMVECTOR calculate_cell_position_vector(tdca* tdca, cell_index cell)
{
    cell_index cells_per_axis = 1 << tdca->lifespace.subdivision_count;
    cell_index cells_per_slice = 1 << (tdca->lifespace.subdivision_count * 2);
    
    real32 x = ((real32) (cell % cells_per_axis)) / (real32) cells_per_axis;
    real32 y = -((real32) (((int32) ((real32) cell / cells_per_axis)) % (cells_per_slice / cells_per_axis)) / (real32) cells_per_axis);
    real32 z = -((real32) ((int32) ((real32) cell / cells_per_slice)) / (real32) cells_per_axis);

    return DirectX::XMVectorSet(x, y, z, 1.0f);
}

void cuda_update_voxels(tdca* tdca)
{
    cell_index alive_cell_counter = 0;
    for(cell_index cell = 0; cell < (real32) tdca->lifespace.cell_count; cell++)
    {
        if(tdca->lifespace.current_cells[cell].state == tdca_cell::ALIVE || tdca->lifespace.current_cells[cell].state == tdca_cell::DYING)
        {
            g_instances[alive_cell_counter].translation = calculate_cell_position_vector(tdca, cell);
            alive_cell_counter++;
        }
    }
    g_dx11_voxel->instance_count = alive_cell_counter;

    hipGraphicsMapResources(1, &g_graphics_resource, 0);

    voxel_instance_transforms* instance_buffer;
    size_t instance_buffer_size;
    hipGraphicsResourceGetMappedPointer((void **) &instance_buffer, &instance_buffer_size, g_graphics_resource);

    hipMemcpy(instance_buffer, g_instances, sizeof(voxel_instance_transforms) * alive_cell_counter, hipMemcpyDeviceToDevice);

    hipGraphicsUnmapResources(1, &g_graphics_resource, 0);

    hipMemcpy(tdca->lifespace.last_cells, tdca->lifespace.current_cells, sizeof(tdca_cell) * tdca->lifespace.cell_count, hipMemcpyDeviceToDevice);
    hipMemset(tdca->lifespace.current_cells, 0, sizeof(tdca_cell) * tdca->lifespace.cell_count);
}

void cuda_init_voxel_render_data()
{
    hipMallocManaged(&g_dx11_voxel, sizeof(voxel_render_data));
}

void cuda_link_instance_buffer()
{
    cudaGraphicsD3D11RegisterResource(&g_graphics_resource, g_dx11_voxel->instance_buffer, 0);
}