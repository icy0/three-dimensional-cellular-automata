#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include "tdca_types.h"

__device__ unsigned int cell_state(tdca* tdca, int cell)
{
    if(cell < 0 || cell > tdca->lifespace.cell_count)
    {
        return 0;
    }
    else
    {
        return tdca->lifespace.last_cells[cell].state;
    }
}

__device__ bool is_in_bounds_relative(tdca* tdca, int origin, int target)
{
    int cells_per_axis = 1 << tdca->lifespace.subdivision_count;
    int cells_per_slice = 1 << (tdca->lifespace.subdivision_count * 2);

    // the row of cells the current cell index resides in
    int row_start_cell_inclusive = origin - (origin % cells_per_axis);
    int row_end_cell_inclusive = row_start_cell_inclusive + cells_per_axis - 1;

    // the vertical slice of cells the current cell resides in
    int slice_start_cell_inclusive = origin - (origin % cells_per_slice);
    int slice_end_cell_inclusive = slice_start_cell_inclusive + cells_per_slice - 1;

    int difference =  target - origin;

    bool cell_is_right =   difference == 1 
                            || difference == -cells_per_axis + 1 
                            || difference == cells_per_axis + 1 
                            || difference == - cells_per_slice + 1 
                            || difference == - cells_per_slice + -cells_per_axis + 1 
                            || difference == - cells_per_slice + cells_per_axis + 1 
                            || difference == + cells_per_slice + 1 
                            || difference == + cells_per_slice + -cells_per_axis + 1 
                            || difference == + cells_per_slice + cells_per_axis + 1;
    
    bool cell_is_left =    difference == -1 
                            || difference == -cells_per_axis - 1 
                            || difference == cells_per_axis - 1 
                            || difference == - cells_per_slice - 1 
                            || difference == - cells_per_slice + -cells_per_axis - 1 
                            || difference == - cells_per_slice + cells_per_axis - 1 
                            || difference == + cells_per_slice - 1 
                            || difference == + cells_per_slice + -cells_per_axis - 1 
                            || difference == + cells_per_slice + cells_per_axis - 1;

    bool cell_is_top =     difference == -cells_per_axis
                            || difference == -cells_per_axis - 1
                            || difference == -cells_per_axis + 1
                            || difference == -cells_per_axis - cells_per_slice
                            || difference == -cells_per_axis - cells_per_slice - 1
                            || difference == -cells_per_axis - cells_per_slice + 1
                            || difference == -cells_per_axis +  cells_per_slice
                            || difference == -cells_per_axis +  cells_per_slice - 1
                            || difference == -cells_per_axis +  cells_per_slice + 1;

    bool cell_is_bottom =  difference == cells_per_axis
                            || difference == cells_per_axis - 1
                            || difference == cells_per_axis + 1
                            || difference == cells_per_axis - cells_per_slice
                            || difference == cells_per_axis - cells_per_slice - 1
                            || difference == cells_per_axis - cells_per_slice + 1
                            || difference == cells_per_axis +  cells_per_slice
                            || difference == cells_per_axis +  cells_per_slice - 1
                            || difference == cells_per_axis +  cells_per_slice + 1;

    if(cell_is_right)
        if ((origin + 1) > row_end_cell_inclusive) return false;
    if(cell_is_left)
        if ((origin - 1) < row_start_cell_inclusive) return false;
    if(cell_is_top)
        if ((origin - cells_per_axis) < slice_start_cell_inclusive) return false;
    if(cell_is_bottom)
        if ((origin + cells_per_axis) > slice_end_cell_inclusive) return false;

    return true;
}

__device__ void count_alive_neighbors_moore(tdca* tdca, int cell, unsigned int* alive_neighbors_count)
{
    int cells_per_axis = 1 << tdca->lifespace.subdivision_count;
    int cells_per_slice = 1 << (tdca->lifespace.subdivision_count * 2);

    if(is_in_bounds_relative(tdca, cell, cell - 1))                                       *alive_neighbors_count += (cell_state(tdca, cell - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + 1))                                       *alive_neighbors_count += (cell_state(tdca, cell + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_axis))                          *alive_neighbors_count += (cell_state(tdca, cell - cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_axis - 1))                      *alive_neighbors_count += (cell_state(tdca, cell - cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_axis + 1))                      *alive_neighbors_count += (cell_state(tdca, cell - cells_per_axis + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_axis))                          *alive_neighbors_count += (cell_state(tdca, cell + cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_axis - 1))                      *alive_neighbors_count += (cell_state(tdca, cell + cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_axis + 1))                      *alive_neighbors_count += (cell_state(tdca, cell + cells_per_axis + 1) > 0 ? 1 : 0);

    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice))                         *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice - 1))                     *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice + 1))                     *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice - cells_per_axis))        *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice - cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice - cells_per_axis - 1))    *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice - cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice - cells_per_axis + 1))    *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice - cells_per_axis + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice + cells_per_axis))        *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice + cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice + cells_per_axis - 1))    *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice + cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice + cells_per_axis + 1))    *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice + cells_per_axis + 1) > 0 ? 1 : 0);

    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice))                         *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice - 1))                     *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice + 1))                     *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice - cells_per_axis))        *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice - cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice - cells_per_axis - 1))    *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice - cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice - cells_per_axis + 1))    *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice - cells_per_axis + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice + cells_per_axis))        *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice + cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice + cells_per_axis - 1))    *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice + cells_per_axis - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice + cells_per_axis + 1))    *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice + cells_per_axis + 1) > 0 ? 1 : 0);
}

__device__ void count_alive_neighbors_von_neumann(tdca* tdca, int cell, unsigned int* alive_neighbors_count)
{
    int cells_per_axis = 1 << tdca->lifespace.subdivision_count;
    int cells_per_slice = 1 << (tdca->lifespace.subdivision_count * 2);

    if(is_in_bounds_relative(tdca, cell, cell - 1))                                       *alive_neighbors_count += (cell_state(tdca, cell - 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + 1))                                       *alive_neighbors_count += (cell_state(tdca, cell + 1) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_axis))                          *alive_neighbors_count += (cell_state(tdca, cell - cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_axis))                          *alive_neighbors_count += (cell_state(tdca, cell + cells_per_axis) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell - cells_per_slice))                         *alive_neighbors_count += (cell_state(tdca, cell - cells_per_slice) > 0 ? 1 : 0);
    if(is_in_bounds_relative(tdca, cell, cell + cells_per_slice))                         *alive_neighbors_count += (cell_state(tdca, cell + cells_per_slice) > 0 ? 1 : 0);
}

__global__ void update_lifespace_partition(tdca* tdca)
{
    unsigned int thread_id = threadIdx.x;
    unsigned int block_id = blockIdx.x;
    unsigned int number_of_threads_per_block = blockDim.x;
    unsigned int number_of_blocks_per_grid = gridDim.x;

    int cells_per_thread = tdca->lifespace.cell_count / (number_of_threads_per_block * number_of_blocks_per_grid);
    
    int starting_cell = (block_id * (cells_per_thread * number_of_threads_per_block)) + (thread_id * cells_per_thread);
    int ending_cell = starting_cell + cells_per_thread - 1;
    
    for(int cell = starting_cell; cell < ending_cell + 1; cell++)
    {
        unsigned int alive_neighbor_count = 0;
        if(tdca->rule.neighborhood == tdca_rule::MOORE)
        {
            count_alive_neighbors_moore(tdca, cell, &alive_neighbor_count); // make device function
        }
        else if(tdca->rule.neighborhood == tdca_rule::VON_NEUMANN)
        {
            count_alive_neighbors_von_neumann(tdca, cell, &alive_neighbor_count); // make device function
        }

        if(cell_state(tdca, cell) == tdca_cell::ALIVE)
        {
            if(tdca->rule.necessary_amounts_of_alive_neighbors_for_surviving[alive_neighbor_count] == 0)
            {
                if(tdca->rule.state_count == 2)
                {
                    tdca->lifespace.current_cells[cell].state = tdca_cell::DEAD;
                    tdca->lifespace.current_cells[cell].lifetime = 0;
                }
                else
                {
                    tdca->lifespace.current_cells[cell].state = tdca_cell::DYING;
                    tdca->lifespace.current_cells[cell].lifetime--;
                }
            }
        }
        else if(cell_state(tdca, cell) == tdca_cell::DYING)
        {
            if(tdca->lifespace.last_cells[cell].lifetime == 0)
            {
                tdca->lifespace.current_cells[cell].state = tdca_cell::DEAD;
            }
            else
            {
                tdca->lifespace.current_cells[cell].lifetime--;
            }
        }
        else if(cell_state(tdca, cell) == tdca_cell::DEAD)
        {
            if(tdca->rule.necessary_amounts_of_alive_neighbors_for_birth[alive_neighbor_count] == 1)
            {
                tdca->lifespace.current_cells[cell].state = tdca_cell::ALIVE;
                tdca->lifespace.current_cells[cell].lifetime = tdca->rule.state_count - 1;
            }
        }
    }
}

void cuda_update_current_buffer(tdca* tdca)
{
    hipMemcpy(tdca->lifespace.current_cells, tdca->lifespace.last_cells, sizeof(tdca_cell) * tdca->lifespace.cell_count, hipMemcpyDefault);

    unsigned int partition_count = 1 << tdca->spacial_partitioning_scheme.subdivision_count;

    update_lifespace_partition<<<partition_count,1024>>>(tdca);
    int error_code = hipDeviceSynchronize();
    int dummy = 42;
}

void cuda_init(tdca** tdca, unsigned int tdca_size_in_bytes)
{
    hipMallocManaged(tdca, tdca_size_in_bytes);
    (*tdca)->lifespace.subdivision_count = 6;
    (*tdca)->lifespace.cell_count = 1 << ((*tdca)->lifespace.subdivision_count * 3);

    hipMallocManaged(&(*tdca)->lifespace.current_cells, (*tdca)->lifespace.cell_count * sizeof(tdca_cell));
    // hipMemset(&tdca->lifespace.current_cells, 0, tdca->lifespace.cell_count * sizeof(tdca_cell));

    hipMallocManaged(&(*tdca)->lifespace.last_cells, (*tdca)->lifespace.cell_count * sizeof(tdca_cell));
    // hipMemset(&tdca->lifespace.last_cells, 0, tdca->lifespace.cell_count * sizeof(tdca_cell));

    (*tdca)->rule.neighborhood = (*tdca)->rule.neighborhood::MOORE;

    (*tdca)->spacial_partitioning_scheme.scheme = (*tdca)->spacial_partitioning_scheme.scheme::BINARY;
    (*tdca)->spacial_partitioning_scheme.subdivision_count = 6;
}

void cuda_free(tdca* tdca)
{
    hipFree(tdca->lifespace.current_cells);
    hipFree(tdca->lifespace.last_cells);
}